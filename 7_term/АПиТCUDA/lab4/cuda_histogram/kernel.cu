#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <vector>
#include <algorithm>
#include <cassert>

#define ARRAY_SIZE 10000000  // Размер массива
#define BIN_COUNT 30000       // Количество корзин для гистограммы

// CUDA kernel для вычисления гистограммы
__global__ void cudaHistogram(const short* data, int* hist, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Используем атомарные операции для корректного подсчета гистограммы
    if (idx < size) {
        atomicAdd(&hist[data[idx]], 1);
    }
}

// Функция для генерации случайных чисел в массиве
void generateRandomData(short* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = rand() % BIN_COUNT;  // Числа от 0 до BIN_COUNT - 1
    }
}

// CPU версия гистограммы
void cpuHistogram(const short* data, int* hist, int size) {
    for (int i = 0; i < size; ++i) {
        hist[data[i]]++;
    }
}

// Функция для вывода гистограммы
void printHistogram(const int* hist, int count) {
    for (int i = 0; i < count; ++i) {
        std::cout << "Count of " << i << ": " << hist[i] << std::endl;
    }
}

// Функция для вывода части массива
void printArray(const short* data, int size, int limit) {
    std::cout << "Array (first " << limit << " elements): ";
    for (int i = 0; i < limit && i < size; ++i) {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;
}

// Сравнение двух гистограмм
void compareHistograms(const int* hist1, const int* hist2) {
    for (int i = 0; i < BIN_COUNT; ++i) {
        assert(hist1[i] == hist2[i] && "Histograms do not match!");
    }
    std::cout << "Histograms match!" << std::endl;
}

int main() {
    // Инициализация
    srand(time(0));

    // Создаем массив данных на CPU
    std::vector<short> data(ARRAY_SIZE);
    generateRandomData(data.data(), ARRAY_SIZE);

    // Гистограммы для CPU и CUDA
    std::vector<int> histCpu(BIN_COUNT, 0);
    std::vector<int> histCuda(BIN_COUNT, 0);

    // Вывод части исходного массива
    printArray(data.data(), ARRAY_SIZE, 100);  // Вывод первых 10 элементов массива

    // --- CPU реализация ---
    auto startCpu = std::chrono::high_resolution_clock::now();
    cpuHistogram(data.data(), histCpu.data(), ARRAY_SIZE);
    auto endCpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpuDuration = endCpu - startCpu;

    std::cout << "CPU histogram completed in: " << cpuDuration.count() << " seconds." << std::endl;

    // --- CUDA реализация ---
    short* d_data;
    int* d_hist;

    // Выделяем память на GPU
    hipMalloc(&d_data, ARRAY_SIZE * sizeof(short));
    hipMalloc(&d_hist, BIN_COUNT * sizeof(int));

    // Копируем данные на GPU
    hipMemcpy(d_data, data.data(), ARRAY_SIZE * sizeof(short), hipMemcpyHostToDevice);
    hipMemset(d_hist, 0, BIN_COUNT * sizeof(int));

    // Запуск CUDA Kernel
    int blockSize = 512;
    int numBlocks = (ARRAY_SIZE + blockSize - 1) / blockSize;

    auto startCuda = std::chrono::high_resolution_clock::now();
    cudaHistogram << <numBlocks, blockSize >> > (d_data, d_hist, ARRAY_SIZE);
    hipDeviceSynchronize();
    auto endCuda = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cudaDuration = endCuda - startCuda;

    // Копируем результаты с GPU на CPU
    hipMemcpy(histCuda.data(), d_hist, BIN_COUNT * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "CUDA histogram completed in: " << cudaDuration.count() << " seconds." << std::endl;

    // Сравниваем результаты
    compareHistograms(histCpu.data(), histCuda.data());

    // Вывод гистограммы
    std::cout << "\nCPU Histogram:" << std::endl;
    printHistogram(histCpu.data(), 10);

    std::cout << "\nCUDA Histogram:" << std::endl;
    printHistogram(histCuda.data(), 10);

    // Освобождаем память
    hipFree(d_data);
    hipFree(d_hist);

    return 0;
}
