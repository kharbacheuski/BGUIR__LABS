﻿#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <cmath> // для fabsf

// Функция для создания матрицы с случайными значениями
std::vector<std::vector<float>> create_random_matrix(int rows, int cols) {
    std::vector<std::vector<float>> matrix(rows, std::vector<float>(cols));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);

    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            matrix[i][j] = dis(gen);

    return matrix;
}

// Ядро CUDA для отражения по вертикали
__global__ void flip_vertically_gpu(float* matrix, float* result, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_size = rows * cols;

    if (idx < total_size) {
        int row = idx / cols;
        int col = idx % cols;
        int new_row = rows - 1 - row;
        result[new_row * cols + col] = matrix[row * cols + col];
    }
}

// Ядро CUDA для отражения по горизонтали
__global__ void flip_horizontally_gpu(float* matrix, float* result, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_size = rows * cols;

    if (idx < total_size) {
        int row = idx / cols;
        int col = idx % cols;
        int new_col = cols - 1 - col;
        result[row * cols + new_col] = matrix[row * cols + col];
    }
}

// Функция для отражения по вертикали на CPU
std::vector<std::vector<float>> flip_vertically(const std::vector<std::vector<float>>& matrix) {
    std::vector<std::vector<float>> flipped(matrix);
    std::reverse(flipped.begin(), flipped.end());
    return flipped;
}

// Функция для отражения по горизонтали на CPU
std::vector<std::vector<float>> flip_horizontally(const std::vector<std::vector<float>>& matrix) {
    std::vector<std::vector<float>> flipped(matrix);
    for (auto& row : flipped)
        std::reverse(row.begin(), row.end());
    return flipped;
}

// Функция для замера времени выполнения на CPU
void measure_cpu(const std::vector<std::vector<float>>& matrix, std::vector<std::vector<float>>& result) {
    auto start = std::chrono::high_resolution_clock::now();

    // Отражение по вертикали
    auto vertically_flipped = flip_vertically(matrix);

    // Отражение по горизонтали
    result = flip_horizontally(vertically_flipped);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;

    std::cout << "CPU Time: " << duration.count() << " seconds" << std::endl;
}

// Функция для замера времени выполнения на GPU
void measure_gpu(float* d_matrix, float* d_temp_matrix, float* d_result_matrix, int rows, int cols) {
    int total_size = rows * cols;
    int threads_per_block = 256;
    int blocks = (total_size + threads_per_block - 1) / threads_per_block;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Отражение по вертикали
    flip_vertically_gpu<<<blocks, threads_per_block>>>(d_matrix, d_temp_matrix, rows, cols);
    hipDeviceSynchronize();

    // Отражение по горизонтали
    flip_horizontally_gpu<<<blocks, threads_per_block>>>(d_temp_matrix, d_result_matrix, rows, cols);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Time: " << milliseconds / 1000.0 << " seconds" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Полное поэлементное сравнение матриц
bool compare_results(const std::vector<std::vector<float>>& cpu_matrix, const std::vector<float>& gpu_matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            if (fabsf(cpu_matrix[i][j] - gpu_matrix[i * cols + j]) > 1e-5) {
                return false; // Результаты различаются
            }
        }
    }
    return true; // Результаты совпадают
}

int main() {
    int rows = 1000, cols = 1000;
    size_t matrix_size = rows * cols * sizeof(float);

    // Создание матрицы на CPU
    auto cpu_matrix = create_random_matrix(rows, cols);

    // Копирование данных в линейный массив для GPU
    std::vector<float> flat_cpu_matrix(rows * cols);
    for (int i = 0; i < rows; ++i)
        std::copy(cpu_matrix[i].begin(), cpu_matrix[i].end(), flat_cpu_matrix.begin() + i * cols);

    // Выделение памяти на GPU
    float* d_matrix;
    float* d_temp_matrix;
    float* d_result_matrix;
    hipMalloc(&d_matrix, matrix_size);
    hipMalloc(&d_temp_matrix, matrix_size);
    hipMalloc(&d_result_matrix, matrix_size);

    // Копирование данных на GPU
    hipMemcpy(d_matrix, flat_cpu_matrix.data(), matrix_size, hipMemcpyHostToDevice);

    // Замер времени на CPU
    std::vector<std::vector<float>> cpu_result(rows, std::vector<float>(cols));
    measure_cpu(cpu_matrix, cpu_result);

    // Замер времени на GPU
    measure_gpu(d_matrix, d_temp_matrix, d_result_matrix, rows, cols);

    // Получение результатов с GPU
    std::vector<float> gpu_result(rows * cols);
    hipMemcpy(gpu_result.data(), d_result_matrix, matrix_size, hipMemcpyDeviceToHost);

    // Сравнение результатов
    if (compare_results(cpu_result, gpu_result, rows, cols)) {
        std::cout << "CPU and GPU results match!" << std::endl;
    } else {
        std::cout << "Results differ!" << std::endl;
    }

    // Освобождение памяти
    hipFree(d_matrix);
    hipFree(d_temp_matrix);
    hipFree(d_result_matrix);

    return 0;
}
