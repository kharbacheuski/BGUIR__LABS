#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""
#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <cmath>

#define MASK_WIDTH 3
#define MASK_RADIUS (MASK_WIDTH / 2)
#define THREADS_PER_BLOCK 32

// Матрица фильтра высокого разрешения H1 CUDA
__constant__ int filter_CUDA[MASK_WIDTH * MASK_WIDTH] = {
    -1, -1, -1,
    -1, 9, -1,
    -1, -1, -1
};

int filter_CPU[MASK_WIDTH * MASK_WIDTH] = {
    -1, -1, -1,
    -1, 9, -1,
    -1, -1, -1
};

// Функция для загрузки PPM изображения
bool loadPPM(const std::string& filename, int& width, int& height, std::vector<unsigned char>& data) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) return false;

    std::string header;
    file >> header;
    if (header != "P6") return false;

    file >> width >> height;
    int maxval;
    file >> maxval;
    file.get();  // Пропустить один байт

    data.resize(width * height * 3);
    file.read(reinterpret_cast<char*>(data.data()), data.size());
    return true;
}

// Функция для сохранения PPM изображения
bool savePPM(const std::string& filename, int width, int height, const std::vector<unsigned char>& data) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) return false;

    file << "P6\n" << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<const char*>(data.data()), data.size());
    return true;
}


// CPU-реализация фильтра
void applyHighPassFilterCPU(const std::vector<unsigned char>& input, std::vector<unsigned char>& output, int width, int height) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            if (x >= MASK_RADIUS && x < width - MASK_RADIUS && y >= MASK_RADIUS && y < height - MASK_RADIUS) {
                for (int c = 0; c < 3; ++c) {  // Для каждого цветового канала
                    int sum = 0;
                    for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                        for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                            // Получение значения пикселя с учетом краевых условий
                            int pixel = input[((y + dy) * width + (x + dx)) * 3 + c];
                            sum += pixel * filter_CPU[(dy + MASK_RADIUS) * MASK_WIDTH + (dx + MASK_RADIUS)];
                        }
                    }
                    sum = std::min(255, std::max(0, sum)); // Ограничение по диапазону
                    output[(y * width + x) * 3 + c] = static_cast<unsigned char>(sum);
                }
            }
        }
    }
}

// CUDA-ядро для фильтрации
__global__ void applyHighPassFilterCUDA(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= MASK_RADIUS && x < width - MASK_RADIUS && y >= MASK_RADIUS && y < height - MASK_RADIUS) {
        for (int c = 0; c < 3; ++c) {  // Для каждого цветового канала
            int sum = 0;
            for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                    int pixel = input[((y + dy) * width + (x + dx)) * 3 + c];
                    sum += pixel * filter_CUDA[(dy + MASK_RADIUS) * MASK_WIDTH + (dx + MASK_RADIUS)];
                }
            }

            sum = min(255, max(0, sum));
            output[(y * width + x) * 3 + c] = static_cast<unsigned char>(sum);
        }
    }
}

// Сравнение результатов CPU и GPU
bool compareResults(const std::vector<unsigned char>& cpu, const std::vector<unsigned char>& gpu, int width, int height) {
    int nots = 0;
    for (int i = 0; i < width * height * 3; ++i) {
        if (std::abs(cpu[i] - gpu[i]) > 1) {
            nots++;
        }
    }
    if (nots > 1) {
        std::cerr << "Несовпадений найдено " << nots << std::endl;
        return false;
    }
    return true;
}

int main() {
    int width, height;
    char *path = "C:\\Users\\kiryl.harbacheuski\\Desktop\\belka.ppm";
    std::vector<unsigned char> image;
    if (!loadPPM(path, width, height, image)) {
        std::cerr << "Не удалось загрузить изображение!" << std::endl;
        return -1;
    }

    std::vector<unsigned char> outputCPU(image.size());
    std::vector<unsigned char> outputGPU(image.size());

    // Применение фильтра на CPU
    auto startCPU = std::chrono::high_resolution_clock::now();
    applyHighPassFilterCPU(image, outputCPU, width, height);
    auto endCPU = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpuTime = endCPU - startCPU;
    std::cout << "Время обработки на CPU: " << cpuTime.count() << " секунд" << std::endl;

    // Применение фильтра на GPU
    unsigned char* d_input, * d_output;
    hipMalloc(&d_input, image.size());
    hipMalloc(&d_output, image.size());
    hipMemcpy(d_input, image.data(), image.size(), hipMemcpyHostToDevice);

    dim3 blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridSize((width + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    auto startGPU = std::chrono::high_resolution_clock::now();
    applyHighPassFilterCUDA << <gridSize, blockSize >> > (d_input, d_output, width, height);
    hipDeviceSynchronize();
    auto endGPU = std::chrono::high_resolution_clock::now();

    hipMemcpy(outputGPU.data(), d_output, image.size(), hipMemcpyDeviceToHost);
    std::chrono::duration<double> gpuTime = endGPU - startGPU;
    std::cout << "Время обработки на GPU: " << gpuTime.count() << " секунд" << std::endl;

    // Сравнение результатов
    if (compareResults(outputCPU, outputGPU, width, height)) {
        std::cout << "Результаты CPU и GPU совпадают." << std::endl;
    }
    else {
        std::cout << "Результаты CPU и GPU НЕ совпадают." << std::endl;
    }

    // Сохранение изображения
    if (!savePPM("output-gpu.ppm", width, height, outputGPU)) {
        std::cerr << "Не удалось сохранить изображение!" << std::endl;
        return -1;
    }

    // Сохранение изображения
    if (!savePPM("output-cpu.ppm", width, height, outputCPU)) {
        std::cerr << "Не удалось сохранить изображение!" << std::endl;
        return -1;
    }


    hipFree(d_input);
    hipFree(d_output);

    return 0;
}