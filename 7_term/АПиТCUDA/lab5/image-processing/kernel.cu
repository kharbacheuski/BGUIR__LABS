#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

#define MASK_WIDTH 3
#define MASK_RADIUS (MASK_WIDTH / 2)
#define THREADS_PER_BLOCK 32

__constant__ int filter_CUDA[MASK_WIDTH * MASK_WIDTH] = {
    -1, -1, -1,
    -1, 9, -1,
    -1, -1, -1
};

int filter_CPU[MASK_WIDTH * MASK_WIDTH] = {
    -1, -1, -1,
    -1, 9, -1,
    -1, -1, -1
};

bool loadImage(const std::string& filename, int& width, int& height, std::vector<unsigned char>& data, bool& isGrayscale) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) return false;

    std::string header;
    file >> header;
    if (header == "P6") {
        isGrayscale = false;
    }
    else if (header == "P5") {
        isGrayscale = true;
    }
    else {
        return false;
    }

    file >> width >> height;
    int maxval;
    file >> maxval;
    file.get();

    int numChannels = isGrayscale ? 1 : 3;
    data.resize(width * height * numChannels);
    file.read(reinterpret_cast<char*>(data.data()), data.size());
    return true;
}

bool saveImage(const std::string& filename, int width, int height, const std::vector<unsigned char>& data, bool isGrayscale) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) return false;

    file << (isGrayscale ? "P5" : "P6") << "\n";
    file << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<const char*>(data.data()), data.size());
    return true;
}

void applyHighPassFilterCPU(const std::vector<unsigned char>& input, std::vector<unsigned char>& output, int width, int height) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            if (x >= MASK_RADIUS && x < width - MASK_RADIUS && y >= MASK_RADIUS && y < height - MASK_RADIUS) {
                for (int c = 0; c < 3; ++c) {
                    int sum = 0;
                    for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                        for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                            int pixel = input[((y + dy) * width + (x + dx)) * 3 + c];
                            sum += pixel * filter_CPU[(dy + MASK_RADIUS) * MASK_WIDTH + (dx + MASK_RADIUS)];
                        }
                    }
                    sum = std::min(255, std::max(0, sum));
                    output[(y * width + x) * 3 + c] = static_cast<unsigned char>(sum);
                }
            }
        }
    }
}

void applyHighPassFilterCPU_Grayscale(const std::vector<unsigned char>& input, std::vector<unsigned char>& output, int width, int height) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            if (x >= MASK_RADIUS && x < width - MASK_RADIUS && y >= MASK_RADIUS && y < height - MASK_RADIUS) {
                int sum = 0;
                for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                    for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                        int pixel = input[(y + dy) * width + (x + dx)];
                        sum += pixel * filter_CPU[(dy + MASK_RADIUS) * MASK_WIDTH + (dx + MASK_RADIUS)];
                    }
                }
                sum = std::min(255, std::max(0, sum));
                output[y * width + x] = static_cast<unsigned char>(sum);
            }
        }
    }
}

__global__ void applyHighPassFilterCUDA(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= MASK_RADIUS && x < width - MASK_RADIUS && y >= MASK_RADIUS && y < height - MASK_RADIUS) {
        for (int c = 0; c < 3; ++c) {
            int sum = 0;
            for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                    int pixel = input[((y + dy) * width + (x + dx)) * 3 + c];
                    sum += pixel * filter_CUDA[(dy + MASK_RADIUS) * MASK_WIDTH + (dx + MASK_RADIUS)];
                }
            }
            sum = min(255, max(0, sum));
            output[(y * width + x) * 3 + c] = static_cast<unsigned char>(sum);
        }
    }
}

__global__ void applyHighPassFilterCUDA_Grayscale(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= MASK_RADIUS && x < width - MASK_RADIUS && y >= MASK_RADIUS && y < height - MASK_RADIUS) {
        int sum = 0;
        for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
            for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                int pixel = input[(y + dy) * width + (x + dx)];
                sum += pixel * filter_CUDA[(dy + MASK_RADIUS) * MASK_WIDTH + (dx + MASK_RADIUS)];
            }
        }
        sum = min(255, max(0, sum));
        output[y * width + x] = static_cast<unsigned char>(sum);
    }
}

int main() {
    int width, height;
    bool isGrayscale;
    std::string path = "C:\\Users\\kiryl.harbacheuski\\Desktop\\belka.pgm";
    std::vector<unsigned char> image;

    if (!loadImage(path, width, height, image, isGrayscale)) {
        std::cerr << "Не удалось загрузить изображение!" << std::endl;
        return -1;
    }

    std::vector<unsigned char> outputCPU(image.size());
    std::vector<unsigned char> outputGPU(image.size());

    auto startCPU = std::chrono::high_resolution_clock::now();
    if (isGrayscale) {
        applyHighPassFilterCPU_Grayscale(image, outputCPU, width, height);
    }
    else {
        applyHighPassFilterCPU(image, outputCPU, width, height);
    }
    auto endCPU = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> durationCPU = endCPU - startCPU;
    std::cout << "Время выполнения на CPU: " << durationCPU.count() << " секунд\n";

    unsigned char* d_input, * d_output;
    hipMalloc(&d_input, image.size());
    hipMalloc(&d_output, image.size());
    hipMemcpy(d_input, image.data(), image.size(), hipMemcpyHostToDevice);

    dim3 blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridSize((width + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    auto startGPU = std::chrono::high_resolution_clock::now();
    if (isGrayscale) {
        applyHighPassFilterCUDA_Grayscale << <gridSize, blockSize >> > (d_input, d_output, width, height);
    }
    else {
        applyHighPassFilterCUDA << <gridSize, blockSize >> > (d_input, d_output, width, height);
    }
    hipDeviceSynchronize();
    auto endGPU = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> durationGPU = endGPU - startGPU;
    std::cout << "Время выполнения на GPU: " << durationGPU.count() << " секунд\n";

    hipMemcpy(outputGPU.data(), d_output, image.size(), hipMemcpyDeviceToHost);

    // Сравнение результатов CPU и GPU
    bool match = true;
    for (size_t i = 0; i < image.size(); ++i) {
        if (outputCPU[i] != outputGPU[i]) {
            match = false;
            std::cout << "Несоответствие в пикселе " << i << ": CPU = " << static_cast<int>(outputCPU[i]) << ", GPU = " << static_cast<int>(outputGPU[i]) << "\n";
            break;
        }
    }
    if (match) {
        std::cout << "Результаты CPU и GPU совпадают.\n";
    }
    else {
        std::cout << "Результаты CPU и GPU не совпадают.\n";
    }

    saveImage("outputCPU", width, height, outputCPU, isGrayscale);
    saveImage("outputGPU", width, height, outputGPU, isGrayscale);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
